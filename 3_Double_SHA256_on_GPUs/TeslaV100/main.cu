#include "hip/hip_runtime.h"
// cd /home/hork/cuda-workspace/CudaSHA256/Debug/files
// time ~/Dropbox/FIIT/APS/Projekt/CpuSHA256/a.out -f ../file-list
// time ../CudaSHA256 -f ../file-list


#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include "sha256.cuh"
#include <dirent.h>
#include <ctype.h>
#include <sys/time.h>

#define FILE_OUTPUT "timing_report.log"

#define N 65536
#define BLOCKSIZE 196
#define M 4294967296/N

void string2ByteArray(char* input, BYTE* output)
{
    uint32_t loop;
    uint32_t i;
    
    loop = 0;
    i = 0;
    
    while(input[loop] != '\0')
    {
        output[i++] = input[loop++];
    }
}

uint32_t LitToBigEndian(uint32_t x)
{
	return (((x>>24) & 0x000000ff) | ((x>>8) & 0x0000ff00) | ((x<<8) & 0x00ff0000) | ((x<<24) & 0xff000000));
}

__global__ void sha256_cuda(JOB ** jobs, uint32_t n, uint32_t j, OUT * outs) {

   uint32_t index = blockIdx.x * blockDim.x + threadIdx.x;
  uint32_t stride = blockDim.x * gridDim.x;
 
  for (uint32_t i = index; i < n; i += stride){
		SHA256_CTX ctx;
		//sha256_init_23(&ctx);
		WORD temp1[8];
		WORD temp2[8];
		jobs[i]->data2[3] = j*n+i;
		sha256_transform_2(&ctx, jobs[i]->data1, temp1);
		sha256_transform_1(&ctx, temp1, jobs[i]->data2, temp2);
		
		uint32_t k;
		for (k = 0; k < 8; k++)
		{
		jobs[i]->temp[k] = temp2[k];
		}
		jobs[i]->temp[8] = 0x80000000;
		for (k = 9; k < 14; k++)
		{
		jobs[i]->temp[k] = 0;
		}
		jobs[i]->temp[15] = 0x00000100;

		sha256_transform_2(&ctx, jobs[i]->temp, jobs[i]->digest);
		
		uint32_t m;
		
		uint32_t Final_Hash[8];
		
		for (m = 0; m < 8; m++)
		{
			Final_Hash[7-m] =  ((jobs[i]->digest[m]>>24) & 0x000000ff) | ((jobs[i]->digest[m]>>8) & 0x0000ff00) | ((jobs[i]->digest[m]<<8) & 0x00ff0000) | ((jobs[i]->digest[m]<<24) & 0xff000000);
		}
		
		int valid = 1;
		for ( m = 0; m < outs->NUM; m ++){
			if(Final_Hash[m] > outs->TARGET[m])
				valid = 0;
		}
		
		if(valid){
			outs->NONCE = jobs[i]->data2[3];
			for (m = 0; m < 8; m++)
			{
				outs->VALID_H[m] =  Final_Hash[m];
			}
		}
	}
}

void pre_sha256() {
	// compy symbols
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dev_k), host_k, sizeof(host_k), 0, hipMemcpyHostToDevice));
}

void runJobs(JOB ** jobs, uint32_t n, uint32_t j, OUT * outs){
	
	uint32_t blockSize = BLOCKSIZE;
	uint32_t numBlocks = (n + blockSize - 1) / blockSize;
	sha256_cuda <<< numBlocks, blockSize >>> (jobs, n, j,outs);
	//sha256_cuda <<< 1, 1 >>> (jobs, n, j, outs);
	//sha256_cuda <<< 1, 16 >>> (jobs, n);
}

JOB * JOB_init(const WORD data1[], const WORD data2[], const WORD H[]) {
	JOB * j;
	checkCudaErrors(hipMallocManaged(&j, sizeof(JOB)));

	for (uint32_t i = 0; i < 16; i++)
	{
		j->data1[i] = data1[i];
	}
	
	for (uint32_t i = 0; i < 16; i++)
	{
		j->data2[i] = data2[i];
	}

	for (uint32_t i = 0; i < 8; i++)
	{
		j->H[i] = H[i];
	}
		
	return j;
}

int main(void)
{
	JOB ** jobs;
	OUT * outs;
	WORD buf1[8];
	uint32_t i,j;

	FILE* fo = fopen(FILE_OUTPUT, "w+");

	////////////////////////////////
	//**BitcoinAtom Block Header**//
	///////////////////////////////

	/*char Version[] = "2000e000";
	char Prev_Hash[] = "000000000000000f5edd17eb45ea50489d171d13e5255fe1ee9e49084eeb65ab";
	char Merk_Hash[] = "f896a21b7213eb5f1b8ba73b277fba850f6ad4eaf9cfa72a2a1b0986e04cdcd5";
	char Time[] = "5F718F4E";
	char Target[] = "1928d33c";*/
	
	////////////////////////////////
	//**BitcoinCash Block Header**//
	///////////////////////////////

	/*char Version[] = "20e00000";
	char Prev_Hash[] = "00000000000000000150983ec2829d878c4b3c65dbb3b2b91bb68e2d5073314d";
	char Merk_Hash[] = "11f642ffaf5fd182bea3c41ce8a635b2b92aa03a7c362171b777a63c5e540f89";
	char Time[] = "5F6F4F19";
	char Target[] = "1802f9c7";*/
	
	////////////////////////////////
	//** BitcoinV Block Header  **//
	///////////////////////////////

	/*char Version[] = "20c00000";
	char Prev_Hash[] = "00000000000000071817e9b8a491790be5835daf63933485d41752513047a94e";
	char Merk_Hash[] = "bf609e249dd579d2fcc20fc4c15686964bc49fa359d056c595984cf758b2b96d";
	char Time[] = "5F760D01";
	char Target[] = "190c1d72";*/
	
	////////////////////////////////
	//**EmbargoCoin Block Header**//
	///////////////////////////////

	/*char Version[] = "00000002";
	char Prev_Hash[] = "0000061e5616fa75619116059b18facaf5e31f661aab1c3548dd3cb061cc9b05";
	char Merk_Hash[] = "46d2deb2ca2340bd17ef5166e24c0475ab1950fc5ef5a90defbe40467ad8afce";
	char Time[] = "5F6F4951";
	char Target[] = "1e0962d9";*/

	////////////////////////////////
	//**EmbargoCoin Block Header**//
	///////////////////////////////
	
	/*char Version[] = "00000002";
	char Prev_Hash[] = "0000061e5616fa75619116059b18facaf5e31f661aab1c3548dd3cb061cc9b05";
	char Merk_Hash[] = "46d2deb2ca2340bd17ef5166e24c0475ab1950fc5ef5a90defbe40467ad8afce";
	char Time[] = "5F6F4951";
	char Target[] = "1e0962d9";*/

	////////////////////////////////
	//** FreiCoin Block Header  **//
	///////////////////////////////
	
	/*char Version[] = "20800000";
	char Prev_Hash[] = "0000000000000116a9ff19c489f2bdba49c387d7da193015ab3aa6a222150573";
	char Merk_Hash[] = "8516eb1f8561b4c954f32bd3f59cae603ba773c6925523b29fad20df9ec84a6d";
	char Time[] = "5F6F474B";
	char Target[] = "1a01e394";*/

	////////////////////////////////
	//** JouleCoin Block Header  **//
	///////////////////////////////
	
	/*char Version[] = "00400004";
	char Prev_Hash[] = "000000000000525e9ed757b108c9c593fb35108fb35f03fd087cfbbc2e71cddb";
	char Merk_Hash[] = "641a7ffbd1a0479428f1d3f803880a86cc7ed914ec97932d780eb7ef9c69ca1b";
	char Time[] = "5F6A3C6F";
	char Target[] = "1b00931b";*/

	////////////////////////////////
	//**Kryptofranc Block Header**//
	///////////////////////////////
	
	/*char Version[] = "20000000";
	char Prev_Hash[] = "0000000000000196d80d750006472b0786fa607114574330a28bc7afe7ef8e70";
	char Merk_Hash[] = "914cfe3a7005c76f808781fafeab874300c514a1a886160e429283906104a3ed";
	char Time[] = "5F71CD79";
	char Target[] = "1a028a1e";*/

	////////////////////////////////
	//** ZetaCoin Block Header **//
	///////////////////////////////
	
	/*char Version[] = "00000002";
	char Prev_Hash[] = "00000000000eb602457fec75d26912c30b8f6740ee26bd53b7a1235dd7847c78";
	char Merk_Hash[] = "3d4874f4a1449e13b303dcd0b74eddd47c1f9b5b8edd2d9d0069163ac56f2fbe";
	char Time[] = "52166E7B";
	char Target[] = "1b176520";*/

	////////////////////////////////
	//**  Bitcoin Block Header  **//
	///////////////////////////////
	
	char Version[] = "37FFE000";
	char Prev_Hash[] = "000000000000000000038973ac554e90636fae2995efa0d1725c00ac4e7dbc35";
	char Merk_Hash[] = "1ef117d88223949d22091e9f6aa01e7f614b9c7e7a609c25808b413639151683";
	char Time[] = "5F715CF2";
	char Target[] = "170E92AA";

	////////////Change to Little Endian///////////
	fprintf(fo, "----------------------We are trying to mine a Bitcoin block----------------------\n");
	fprintf(fo, "*Block header information:\n");
	fprintf(fo, "	+Version : %s\n", Version);
	fprintf(fo, "	+Previous Hash : %s\n", Prev_Hash);
	fprintf(fo, "	+Merkle Hash : %s\n", Merk_Hash);
	fprintf(fo, "	+Timestemp : %s\n", Time);
	fprintf(fo, "	+Target : %s\n", Target);

	//Version
	uint32_t Version_HEX = (uint32_t)strtol(Version, NULL, 16);
	uint32_t Version_LitEndian = LitToBigEndian(Version_HEX);
	
	//Previous hash
	
	uint32_t  Prev_Hash_Int[8];
	char Prev_Hash_temp[8];
	uint32_t  Prev_Hash_Counter = 0;
	uint32_t  Prev_Hash_LitEndian[8];
	
	for (i=0;i<8;i++){
		for(j=i*8;j<i*8+8;j++){
			Prev_Hash_temp[Prev_Hash_Counter] = Prev_Hash[j];
			Prev_Hash_Counter++;
		}
		Prev_Hash_Counter=0;
		Prev_Hash_Int[i] = (uint32_t)strtol(Prev_Hash_temp, NULL, 16);
		Prev_Hash_LitEndian[7-i] = LitToBigEndian(Prev_Hash_Int[i]);
	}
	
	//Merkle hash
	
	uint32_t  Merk_Hash_Int[8];
	char Merk_Hash_temp[8];
	uint32_t  Merk_Hash_Counter = 0;
	uint32_t  Merk_Hash_LitEndian[8];
	
	for (i=0;i<8;i++){
		for(j=i*8;j<i*8+8;j++){
			Merk_Hash_temp[Merk_Hash_Counter] = Merk_Hash[j];
			Merk_Hash_Counter++;
		}
		Merk_Hash_Counter=0;
		Merk_Hash_Int[i] = (uint32_t)strtol(Merk_Hash_temp, NULL, 16);
		Merk_Hash_LitEndian[7-i] = LitToBigEndian(Merk_Hash_Int[i]);
	}

	//Timestamp
	uint32_t Time_HEX = (uint32_t)strtol(Time, NULL, 16);
	uint32_t Time_LitEndian = LitToBigEndian(Time_HEX);
	
	//Target
	uint32_t Target_HEX = (uint32_t)strtol(Target, NULL, 16);
	uint32_t Target_LitEndian = LitToBigEndian(Target_HEX);
	
	uint32_t nbit1 = (Target_HEX >> 24)&0x000000ff;
	uint32_t nbit2 = (Target_HEX >> 16)&0x000000ff;
	uint32_t nbit3 = (Target_HEX >> 8)&0x000000ff;
	uint32_t nbit4 = (Target_HEX)&0x000000ff;

	uint32_t target_8b[32];
	for( i = 0; i < 32; i++){
		if(i == (32 - nbit1 + 2)) {
			target_8b[i] = nbit4;
		}
		else if(i == (32 - nbit1 + 1)) {
			target_8b[i] = nbit3;
		}
		else if(i == (32 - nbit1)) {
			target_8b[i] = nbit2;
		}
		else {
			target_8b[i] = 0;
		}
	}
	
	uint32_t Target_32b[8];
	
	for( i = 0; i < 8; i++){
		Target_32b[i] = (target_8b[i*4]<<24)|(target_8b[i*4 + 1] << 16)|(target_8b[i*4 + 2] << 8)|(target_8b[i*4 + 3]);
	}
	
	int num_int = (32 - nbit1+3) / 4;
	
	checkCudaErrors(hipMallocManaged(&outs, sizeof(OUT)));
	outs->NUM = num_int;

	
	for( i = 0; i < 8; i++){
		outs->TARGET[i] = Target_32b[i];
	}

	
	fprintf(fo, "*Start to mine........\n");
	clock_t start, end;
	double cpu_time_used;
	int GPU_N;
	start = clock();
	checkCudaErrors(hipGetDeviceCount(&GPU_N));
	checkCudaErrors(hipSetDevice(GPU_N-2));
	//sha256_transform_0(&ctx1, Word1, buf1);
	
	checkCudaErrors(hipMallocManaged(&jobs, N * sizeof(JOB *)));

	for (i=0; i < N; ++i){	
			WORD Word1[16] = {Version_LitEndian, Prev_Hash_LitEndian[0], Prev_Hash_LitEndian[1], Prev_Hash_LitEndian[2], Prev_Hash_LitEndian[3], Prev_Hash_LitEndian[4], Prev_Hash_LitEndian[5], Prev_Hash_LitEndian[6], Prev_Hash_LitEndian[7], Merk_Hash_LitEndian[0], Merk_Hash_LitEndian[1], Merk_Hash_LitEndian[2], Merk_Hash_LitEndian[3], Merk_Hash_LitEndian[4], Merk_Hash_LitEndian[5], Merk_Hash_LitEndian[6]};
			WORD Word2[16] = {Merk_Hash_LitEndian[7], Time_LitEndian, Target_LitEndian, 0x00000000, 0x80000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000280};	
			jobs[i] 	   = JOB_init(Word1, Word2, buf1);
	}

	for(j = 0; j <M; ++j){
		pre_sha256();
		runJobs(jobs, N, j, outs);
	}
	hipDeviceSynchronize();	

	end = clock();
	cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;

	fprintf(fo, "*Execution Time of 2^32 hashes on GPU : %f seconds\n", cpu_time_used);

	fprintf(fo, "*A found nonce:%08x\n", outs->NONCE);

	fprintf(fo, "*A valid hash: ");

	for (i = 0; i < 8; i++)
		{
			fprintf(fo, "%08x",outs->VALID_H[i]);
		}
	fprintf(fo, "\n");
	hipDeviceReset();


	return 0;
}	
	
